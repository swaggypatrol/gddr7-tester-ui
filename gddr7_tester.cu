#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>

#ifndef DEFAULT_FRACTION
#define DEFAULT_FRACTION 0.80
#endif

#define CUDA_CHECK(x) do { \
  hipError_t _e = (x); \
  if (_e != hipSuccess) { \
    fprintf(stderr, "CUDA error '%s' at %s:%d: %s\n", #x, __FILE__, __LINE__, hipGetErrorString(_e)); \
    std::exit(1); \
  } \
} while(0)

__device__ __forceinline__ uint32_t mix32(uint64_t x) {
  x ^= x >> 33; x *= 0xff51afd7ed558ccdULL;
  x ^= x >> 33; x *= 0xc4ceb9fe1a85ec53ULL;
  x ^= x >> 33;
  return static_cast<uint32_t>(x ^ (x >> 32));
}

// ====== Access order mapping (ensures a permutation) ======
struct MapParams {
  unsigned long long step2; // stride-64KiB step (coprime with n_vec)
  unsigned long long step3; // stride-128KiB step (coprime with n_vec)
  unsigned long long step5; // multiplicative permutation step (coprime with n_vec)
  unsigned int block;       // block-xor size in uint4 lanes, recommend 256 (4KiB)
};

__device__ __forceinline__
size_t map_index_perm(size_t i, size_t n_vec, int mode, const MapParams p)
{
  if (mode == 1) {
    // linear: natural order
    return i;
  } else if (mode == 2) {
    // stride ~64KiB: m = i * step (mod n); step coprime with n => permutation
    return (i * p.step2) % n_vec;
  } else if (mode == 3) {
    // stride ~128KiB
    return (i * p.step3) % n_vec;
  } else if (mode == 4) {
    // block-xor (4KiB): flip bits within full blocks; tail blocks use identity mapping
    const unsigned long long B = p.block;
    unsigned long long full = (n_vec / B) * B;  // coverage of full blocks
    if (i < full) {
      unsigned long long g = i / B, r = i % B;
      unsigned long long r2 = r ^ (B >> 1);     // swap half-block
      return g * B + r2;
    } else {
      return i; // tail block: unchanged to preserve one-to-one mapping
    }
  } else {
    // multiplicative permutation (pseudo-random): odd step5 coprime with n
    return (i * p.step5) % n_vec;
  }
}

// ====== Vectorized kernels ======
__global__ void init_pattern_vec(uint4* __restrict__ data4, size_t n_vec, uint32_t seed) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = (size_t)blockDim.x * gridDim.x;
  for (size_t i = tid; i < n_vec; i += stride) {
    size_t base = i * 4ULL;
    uint4 v;
    v.x = mix32((uint64_t)(base + 0) ^ seed);
    v.y = mix32((uint64_t)(base + 1) ^ seed);
    v.z = mix32((uint64_t)(base + 2) ^ seed);
    v.w = mix32((uint64_t)(base + 3) ^ seed);
    data4[i] = v;
  }
}

__global__ void verify_and_flip_vec(uint4* __restrict__ data4, size_t n_vec,
                                    uint32_t expect_seed, uint32_t next_seed,
                                    unsigned long long* __restrict__ errcount,
                                    int mode, MapParams mp) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = (size_t)blockDim.x * gridDim.x;

  for (size_t i = tid; i < n_vec; i += stride) {
    size_t m = map_index_perm(i, n_vec, mode, mp);  // only changes access order; data locations unchanged
    size_t base = m * 4ULL;
    uint4 v = data4[m];

    uint32_t e0 = mix32((uint64_t)(base + 0) ^ expect_seed);
    uint32_t e1 = mix32((uint64_t)(base + 1) ^ expect_seed);
    uint32_t e2 = mix32((uint64_t)(base + 2) ^ expect_seed);
    uint32_t e3 = mix32((uint64_t)(base + 3) ^ expect_seed);

    if (v.x != e0) atomicAdd(errcount, 1ULL);
    if (v.y != e1) atomicAdd(errcount, 1ULL);
    if (v.z != e2) atomicAdd(errcount, 1ULL);
    if (v.w != e3) atomicAdd(errcount, 1ULL);

    v.x = mix32((uint64_t)(base + 0) ^ next_seed);
    v.y = mix32((uint64_t)(base + 1) ^ next_seed);
    v.z = mix32((uint64_t)(base + 2) ^ next_seed);
    v.w = mix32((uint64_t)(base + 3) ^ next_seed);

    data4[m] = v;
  }
}

// ====== Host: pick coprime steps ======
static inline unsigned long long gcd_ull(unsigned long long a, unsigned long long b) {
  while (b) { unsigned long long t = a % b; a = b; b = t; }
  return a;
}

static inline unsigned long long pick_coprime_step(unsigned long long n, unsigned long long target) {
  if (n <= 1) return 1;
  if (target == 0) target = 1;
  // force odd and search near target for a value coprime with n
  if ((target & 1ULL) == 0) target += 1;
  unsigned long long s = target, delta = 0;
  for (unsigned tries = 0; tries < 100000; ++tries) {
    if (gcd_ull(n, s) == 1ULL) return s;
    // alternate +2 / -2 while expanding outward
    delta += 2;
    unsigned long long up = s + delta;
    if (up > 1 && gcd_ull(n, up) == 1ULL) return up;
    if (s > delta) {
      unsigned long long dn = s - delta;
      if (gcd_ull(n, dn) == 1ULL) return dn;
    }
  }
  // fall back to 1 if no coprime found (extreme case)
  return 1ULL;
}

static inline double toGiB(size_t bytes) {
  return (double)bytes / (1024.0 * 1024.0 * 1024.0);
}

int main(int argc, char* argv[]) {
  double fraction = (argc >= 2) ? std::atof(argv[1]) : DEFAULT_FRACTION;
  if (fraction <= 0.0 || fraction > 0.90) fraction = DEFAULT_FRACTION;
  int chunk_iters = (argc >= 3) ? std::atoi(argv[2]) : 100;
  if (chunk_iters <= 0) chunk_iters = 100;

  CUDA_CHECK(hipSetDevice(0));
  hipDeviceProp_t prop{};
  CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

  size_t freeB = 0, totalB = 0;
  CUDA_CHECK(hipMemGetInfo(&freeB, &totalB));

  size_t targetB = (size_t)(freeB * fraction);
  size_t n_elems = targetB / sizeof(uint32_t);
  n_elems -= (n_elems & 3ULL); // align to 16B
  targetB  = n_elems * sizeof(uint32_t);
  size_t n_vec = n_elems / 4;

  int eccEnabled = 0;
  if (hipDeviceGetAttribute(&eccEnabled, hipDeviceAttributeEccEnabled, 0) != hipSuccess) eccEnabled = 0;

  printf("Device: %s (cc %d.%d, %d SMs)\n", prop.name, prop.major, prop.minor, prop.multiProcessorCount);
  printf("Global Mem: %.2f GiB free / %.2f GiB total\n", toGiB(freeB), toGiB(totalB));
  printf("ECC enabled: %s\n", eccEnabled ? "Yes" : "No");
  printf("Allocated/Tested: %.2f GiB (fraction=%.2f)\n", toGiB(targetB), fraction);
  if (n_vec == 0) { printf("Nothing to test.\n"); return 0; }

  // choose steps that are coprime with n
  MapParams mp{};
  mp.block = 256; // 4KiB per block (256 * 16B)
  mp.step2 = pick_coprime_step(n_vec, (64ULL * 1024) / 16ULL);   // ~64KiB / 16B
  mp.step3 = pick_coprime_step(n_vec, (128ULL * 1024) / 16ULL);  // ~128KiB / 16B
  mp.step5 = pick_coprime_step(n_vec, 2654435761ULL);            // coprime neighbor of permutation constant
  printf("Permutation steps: mode2=%llu, mode3=%llu, mode5=%llu (n_vec=%llu)\n",
         (unsigned long long)mp.step2, (unsigned long long)mp.step3, (unsigned long long)mp.step5,
         (unsigned long long)n_vec);

  uint32_t* d_data = nullptr;
  unsigned long long* d_err = nullptr;
  CUDA_CHECK(hipMalloc(&d_data, targetB));
  CUDA_CHECK(hipMalloc(&d_err, sizeof(unsigned long long)));
  CUDA_CHECK(hipMemset(d_err, 0, sizeof(unsigned long long)));

  const int threads = 256;
  int blocks = prop.multiProcessorCount * 32;
  if (blocks > 65535) blocks = 65535;

  const uint32_t seedA = 0x12345678u;
  const uint32_t seedB = 0xDEADBEEFu;

  printf("Initializing pattern A...\n");
  init_pattern_vec<<<blocks, threads>>>((uint4*)d_data, n_vec, seedA);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  hipEvent_t t0, t1;
  CUDA_CHECK(hipEventCreate(&t0));
  CUDA_CHECK(hipEventCreate(&t1));

  printf("Loop forever. Modes cycle as 1→2→3→4→5\n");
  const double tested_bytes = (double)(n_vec * sizeof(uint4));
  unsigned long long prev_err = 0;
  const int MODE_CNT = 5;

  for (unsigned long long chunk = 1;; ++chunk) {
    int mode = (int)((chunk - 1) % MODE_CNT) + 1;

    CUDA_CHECK(hipEventRecord(t0));
    for (int i = 0; i < chunk_iters; ++i) {
      verify_and_flip_vec<<<blocks, threads>>>((uint4*)d_data, n_vec, seedA, seedB, d_err, mode, mp);
      CUDA_CHECK(hipGetLastError());
      verify_and_flip_vec<<<blocks, threads>>>((uint4*)d_data, n_vec, seedB, seedA, d_err, mode, mp);
      CUDA_CHECK(hipGetLastError());
    }
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(t1));
    CUDA_CHECK(hipEventSynchronize(t1));

    float ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&ms, t0, t1));

    unsigned long long total_err = 0;
    CUDA_CHECK(hipMemcpy(&total_err, d_err, sizeof(total_err), hipMemcpyDeviceToHost));
    unsigned long long delta_err = total_err - prev_err;
    prev_err = total_err;

    const double bytes_processed = (double)chunk_iters * 2.0 * 2.0 * tested_bytes;
    const double gbps = (bytes_processed / (ms / 1000.0)) / (1024.0 * 1024.0 * 1024.0);

    printf("[Chunk %llu | Mode %d] Time: %.2f ms | Bandwidth: %.2f GB/s | New errors: %llu | Total errors: %llu\n",
           chunk, mode, ms, gbps, delta_err, total_err);
    fflush(stdout);
  }

  return 0;
}
